#include "hip/hip_runtime.h"
#include "geometry.cuh"
#include "curl.cuh"
#include "constants.cuh"
#include "dataacc.cuh"
#include <cmath>
#include <string>
#include <iterator>
#include <hip/hip_fp16.h>

__global__ void update_Hz(float *Ex, float *Ey, float *Hz,float*mu,int xmax,int ymax,float xstep,float tstep, float speedoflight)
{
	// Curl
	float CurlEz;
	int i = xmax * blockIdx.y * blockDim.y + xmax * threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
	if(i < xmax * ymax) {

			// Curl
	if (blockIdx.x * blockDim.x + threadIdx.x == xmax-1 && blockIdx.y * blockDim.y + threadIdx.y == ymax-1)
	{
		CurlEz=( (0 - Ey[i]) - (0 - Ex[i]) ) / xstep;
	}

	else if (blockIdx.x * blockDim.x + threadIdx.x == xmax-1 && blockIdx.y * blockDim.y + threadIdx.y != ymax-1) {
		CurlEz =  ( ( 0 - Ey[i] ) - (0 - Ex[i] ) ) /xstep;
	}

	else {
		CurlEz = ( (Ey[i + 1] - Ey[i]) - (Ex[i + xmax] - Ex[i]) ) / xstep;
	}

	//Update
	__syncthreads();
	
	Hz[i] = Hz[i] + (-speedoflight*tstep / mu[i]) * CurlEz; 

	}



}

__global__ void update_Dx(float *Hz, float *Dx,int xmax,int ymax,float xstep,float tstep)
{
	// Curl
	float CurlHx;
	unsigned int i = xmax * blockIdx.y * blockDim.y + xmax * threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
	if(i < xmax * ymax) {
			if(blockIdx.y == 0 && threadIdx.y == 0) {
		CurlHx = (Hz[i] - 0) / xstep;
	}
	
	else {
		CurlHx = (Hz[i] - Hz[i - xmax]) / xstep;
	}

	// Dx Update

	__syncthreads();

	Dx[i] = Dx[i] + (speedoflight * tstep) * CurlHx;


	}


}


__global__ void update_Dy(float *Hz, float *Dy,int xmax,int ymax,float xstep,float tstep)
{
	// Curl
	float CurlHy;
	unsigned int i = xmax * blockIdx.y * blockDim.y + xmax * threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

	if(i < xmax * ymax) {
		// Curl
	if(blockIdx.x == 0 && threadIdx.x == 0) {
		CurlHy = -(Hz[i] - 0);
	}




	else {
		CurlHy = -(Hz[i] - Hz[i - 1]) / xstep;
	}

	// Dy Update
	__syncthreads();
	Dy[i] = Dy[i] + (speedoflight * tstep) * CurlHy;

	}
	
}

__global__ void update_E(float *Dx, float *Dy, float *Ex, float *Ey, float* eps,int xmax,int ymax)
{
	unsigned int i = xmax * blockIdx.y * blockDim.y + xmax * threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

	if(i < xmax*ymax) {
		Ex[i] = Dx[i] / eps[i];
		Ey[i] = Dy[i] / eps[i];
	}
	

}

__global__ void source(float *Hz,float freq, float maxtime,float timestep,int qtime,int xmax,int ymax)
{		
		if (blockIdx.x == 16 && blockIdx.y  == 16 && threadIdx.x > 11 && threadIdx.x < 21 && threadIdx.y < 21 && threadIdx.y > 11) {
			Hz[xmax * blockIdx.y * blockDim.y + xmax * threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x] += 1 * sin(freq * timestep * qtime) * (1-exp(-qtime / 5));
		}
}


int main()
{
	// Geometry Defn and Other Definitions

	float xdim =16.0;
	float ydim = 16.0;
	int resolution = 64;
	float eps_si = pow(3.47,2);
	int maxtime = 2500;

	int xsize = xdim * resolution;
	float freq = 193.41449e12 * 7;
	float xstep = 1e-6 / resolution;
	float courantnumber = 0.5;
	float timestep = courantnumber * xstep / speedoflight;

	// Create Geometry Object

	geometry lattice {resolution,xdim, ydim,timestep};

	// Create E-H field grid and set the simulation functions

	curl Grid2D{lattice};
	
	// Main Loop
	
	int frame = 0;

	float *dE_x,*dE_y;
	float *dD_x,*dD_y;

	float *dH_z;

	float *dmu;
	float *deps;

	int *dxmax;
	int *dymax;

	float *dxstep;
	float *dtstep;
	float *dfreq;
	float *dmaxtime;
	float *dtimestep;
	float *mu;
	float *eps;

	float *E_x, *E_y, *H_z;
	float *D_x, *D_y;

	E_x = (float *)malloc(xsize*xsize * sizeof(float));
	E_y = (float *)malloc(xsize*xsize * sizeof(float));
	H_z = (float *)malloc(xsize*xsize * sizeof(float));
	D_x = (float *)malloc(xsize*xsize * sizeof(float));
	D_y = (float *)malloc(xsize*xsize * sizeof(float));
	mu = (float *)malloc(xsize*xsize * sizeof(float));
	eps = (float *)malloc(xsize*xsize * sizeof(float));
	D_y = (float *)malloc(xsize*xsize * sizeof(float));



	for (int i = 0; i < xsize*xsize; i++)
    	{
        E_x[i] = 0;
        E_y[i] = 0;
		H_z[i] = 0;
		eps[i] = 13;
		mu[i] = 1;
    	}

	hipMalloc((void **)&dE_x, xsize*xsize * sizeof(float));
    hipMalloc((void **)&dE_y, xsize*xsize * sizeof(float));

	hipMalloc((void **)&dD_x, xsize*xsize * sizeof(float));
    hipMalloc((void **)&dD_y, xsize*xsize * sizeof(float));

    hipMalloc((void **)&dH_z, xsize*xsize * sizeof(float));

    hipMalloc((void **)&dmu, xsize*xsize * sizeof(float));
    hipMalloc((void **)&deps, xsize*xsize * sizeof(float));

	hipMalloc((void **)&dymax,sizeof(int));
    hipMalloc((void **)&dxmax,sizeof(int));

	hipMalloc((void **)&dxstep,sizeof(float));
    hipMalloc((void **)&dtstep,sizeof(float));
	hipMalloc((void **)&dmaxtime,sizeof(float));
	hipMalloc((void **)&dtimestep,sizeof(float));
	hipMalloc((void **)&dfreq,sizeof(float));

	hipMemcpy(dE_x, E_x, xsize*xsize* sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dE_y, E_y, xsize*xsize * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(dD_x, D_x, xsize*xsize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dD_y, D_y, xsize*xsize * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dH_z, H_z, xsize*xsize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dmu, mu, xsize*xsize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deps, eps, xsize*xsize * sizeof(float), hipMemcpyHostToDevice);
	


	int xmax = 1024;
	int ymax = 1024;

	hipMemcpy(dxmax, &xmax, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dymax, &ymax,sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dxstep, &xstep,sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dtstep, &timestep,sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dmaxtime, &maxtime, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dfreq, &freq, sizeof(float), hipMemcpyHostToDevice);

	int BLOCK_SIZE=32;
    unsigned int grid_rows = (xmax + BLOCK_SIZE - 1) / BLOCK_SIZE + 1;
    unsigned int grid_cols = (ymax + BLOCK_SIZE - 1) / BLOCK_SIZE + 1;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);



	for (int qtime = 0; qtime < maxtime; qtime++) {

		update_Hz<<<dimGrid, dimBlock>>>(dE_x, dE_y, dH_z, dmu,xmax,ymax,xstep,timestep,speedoflight);
		hipDeviceSynchronize();
		update_Dx<<<dimGrid, dimBlock>>>(dH_z, dD_x,xmax,ymax,xstep,timestep);
		hipDeviceSynchronize();
		update_Dy<<<dimGrid, dimBlock>>>(dH_z, dD_y,xmax,ymax,xstep,timestep);
		hipDeviceSynchronize();
		source<<<dimGrid, dimBlock>>>(dH_z,freq, maxtime,timestep,qtime,xmax,ymax);
		hipDeviceSynchronize();
		update_E<<<dimGrid, dimBlock>>>(dD_x, dD_y, dE_x, dE_y, deps,xmax,ymax);
		hipDeviceSynchronize();

		if (qtime % 10 == 0) {
			hipDeviceSynchronize();
    		hipMemcpy(H_z, dH_z, 1024 * 1024 * sizeof(float), hipMemcpyDeviceToHost);

			string name = "outputs/Hz";
			name.append(to_string(frame));
			string exten = ".dat";


			std::ofstream output_file(name+exten);
			std::ostream_iterator<float> output_iterator(output_file, " ");
			std::copy(H_z, H_z +xsize*xsize, output_iterator);
			frame += 1;
			
	}
			hipDeviceSynchronize();
		}

	}





	
